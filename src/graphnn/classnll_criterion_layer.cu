#include "hip/hip_runtime.h"
#include "classnll_criterion_layer.h"
#include <cmath>
#include "cuda_helper.h"

template<typename Dtype>
__global__ void LogLossKernel(Dtype* dst, Dtype* pred, 
                              int* row_ptr, int* col_idx, Dtype* val, 
                              int nnz, int n_rows, int n_cols)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < nnz)
    {
        int row = get_sp_row_idx(i, row_ptr, n_rows);
        dst[i] = cuda_log(pred[row * n_cols + col_idx[i]]) * val[i];
    }
}

template<typename Dtype>
Dtype GetLogLoss(DenseMat<GPU, Dtype>& pred, SparseMat<GPU, Dtype>& label, DenseMat<GPU, Dtype>& buf)
{        
        int thread_num = min(c_uCudaThreadNum, label.data->nnz);
        int blocksPerGrid = (label.data->nnz + thread_num - 1) / thread_num;
        LogLossKernel <<< blocksPerGrid, thread_num >>> (buf.data, pred.data, 
                                                         label.data->ptr, label.data->col_idx, label.data->val,
                                                         label.data->nnz, pred.rows, pred.cols); 
        Dtype loss = buf.Asum();
        return loss; 
}

template<typename Dtype>
Dtype GetLogLoss(DenseMat<CPU, Dtype>& pred, SparseMat<CPU, Dtype>& label, DenseMat<CPU, Dtype>& buf)
{
        assert(pred.rows == buf.rows);
        Dtype loss = 0.0;
        for (size_t i = 0; i < label.rows; ++i)
        {
            for (int k = label.data->ptr[i]; k < label.data->ptr[i + 1]; ++k)
                loss -= log(pred.data[label.cols * i + label.data->col_idx[k]]) * label.data->val[k];
        }
        return loss;
} 

template class ClassNLLCriterionLayer<CPU, float>;
template class ClassNLLCriterionLayer<CPU, double>;
template class ClassNLLCriterionLayer<GPU, float>;
template class ClassNLLCriterionLayer<GPU, double>;