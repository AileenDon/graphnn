#include "hip/hip_runtime.h"
#include "relu_layer.h"
#include "dense_matrix.h"
#include "cuda_helper.h"
#include "cuda_unary_kernel.cuh"
#include "sparse_matrix.h"
#include <hip/hip_runtime.h>

#define min(x, y) (x < y ? x : y)

// =========================================== relu layer ================================================
template<typename Dtype>
void ReLULayer<GPU, Dtype>::Act(DenseMat<GPU, Dtype>& prev_out, DenseMat<GPU, Dtype>& cur_out)
{
    UnaryOp(cur_out.data, prev_out.data, prev_out.count, UnaryReLU<Dtype>(), cur_out.streamid);
}

template<typename Dtype>
__global__ void ReLUDerivKernel(Dtype *d, Dtype *c, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        d[i] = c[i] > 0 ? d[i] : 0;
    }
}

template<typename Dtype>
void ReLULayer<GPU, Dtype>::Derivative(DenseMat<GPU, Dtype>& dst, DenseMat<GPU, Dtype>& prev_output, 
                            DenseMat<GPU, Dtype>& cur_output, DenseMat<GPU, Dtype>& cur_grad)
{
    dst.CopyFrom(cur_grad);
    int thread_num = min(c_uCudaThreadNum, dst.count);    
    int blocksPerGrid = (dst.count + thread_num - 1) / thread_num;
    ReLUDerivKernel <<< blocksPerGrid, thread_num, 0, GPUHandle::streams[dst.streamid] >>>(dst.data, cur_output.data, dst.count);
}

template class ReLULayer<GPU, float>;
template class ReLULayer<GPU, double>;